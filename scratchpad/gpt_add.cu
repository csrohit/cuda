
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024   // Matrix size

// CUDA kernel for matrix addition
__global__ void matrixAdd(float* A, float* B, float* C, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        C[row * size + col] = A[row * size + col] + B[row * size + col];
    }
}

int main() {
    // Declare host matrices
    float* h_A;
    float* h_B;
    float* h_C;

    // Declare device matrices
    float* d_A;
    float* d_B;
    float* d_C;

    int size = N * N * sizeof(float);

    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // Initialize host matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy host matrices to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockDim(16, 16);  // Number of threads in each block
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);  // Number of blocks

    // Launch kernel
    matrixAdd<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_C[i * N + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

